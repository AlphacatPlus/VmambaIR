/******************************************************************************
 * Copyright (c) 2023, Tri Dao.
 ******************************************************************************/
#include "selective_scan_fwd_kernel_nrow.cuh"

template void selective_scan_fwd_cuda<1, float, float>(SSMParamsBase &params, hipStream_t stream);
template void selective_scan_fwd_cuda<1, at::Half, float>(SSMParamsBase &params, hipStream_t stream);
template void selective_scan_fwd_cuda<1, at::BFloat16, float>(SSMParamsBase &params, hipStream_t stream);


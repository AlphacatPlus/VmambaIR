/******************************************************************************
 * Copyright (c) 2023, Tri Dao.
 ******************************************************************************/
#include "selective_scan_bwd_kernel_nrow.cuh"

template void selective_scan_bwd_cuda<3, float, float>(SSMParamsBwd &params, hipStream_t stream);
template void selective_scan_bwd_cuda<3, at::Half, float>(SSMParamsBwd &params, hipStream_t stream);
template void selective_scan_bwd_cuda<3, at::BFloat16, float>(SSMParamsBwd &params, hipStream_t stream);

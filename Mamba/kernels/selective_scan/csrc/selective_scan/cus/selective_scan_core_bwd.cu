/******************************************************************************
 * Copyright (c) 2023, Tri Dao.
 ******************************************************************************/
#include "selective_scan_bwd_kernel.cuh"

template void selective_scan_bwd_cuda<1, float, float>(SSMParamsBwd &params, hipStream_t stream);
template void selective_scan_bwd_cuda<1, at::Half, float>(SSMParamsBwd &params, hipStream_t stream);
template void selective_scan_bwd_cuda<1, at::BFloat16, float>(SSMParamsBwd &params, hipStream_t stream);

